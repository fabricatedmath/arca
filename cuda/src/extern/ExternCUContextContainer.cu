#include "ExternCUContextContainer.cuh"

extern "C" {
    CUContextContainer* cuContextContainerNew() {
        return new CUContextContainer();
    }

    hipCtx_t* cuContextContainerGetCtx(CUContextContainer* cuContextContainer) {
        return cuContextContainer->getCtx();
    }

    void cuContextContainerDelete(CUContextContainer* cuContextContainer) {
        delete cuContextContainer;
    }
}