#include "hip/hip_runtime.h"
#include "NvrtcContainer.cuh"

#include <iostream>

#include <hip/hiprtc.h>

#define CUDA_DRIVER_API

#include <hip/hip_runtime_api.h>
#include <chrono>
using namespace std::chrono;

using namespace std;

#define NVRTC_SAFE_CALL(Name, x)                                \
  do {                                                          \
    hiprtcResult result = x;                                     \
    if (result != HIPRTC_SUCCESS) {                              \
      std::cerr << "\nerror: " << Name << " failed with error " \
                << hiprtcGetErrorString(result);                 \
      exit(1);                                                  \
    }                                                           \
  } while (0)

NvrtcContainer::NvrtcContainer(CUContextContainer* cuContextContainer) : hModule(0), hKernel(0) {
    hipCtxSetCurrent(*cuContextContainer->getCtx());
}

void NvrtcContainer::init() {
    checkCudaErrors( hipInit(0) );
    hipSetDevice(0);
}

bool NvrtcContainer::compile(const char* str, const int strlen) {
    string progStr(str,strlen);
    auto t0 = steady_clock::now();
    hiprtcProgram prog;
    NVRTC_SAFE_CALL("hiprtcCreateProgram", hiprtcCreateProgram(&prog, progStr.c_str(), "device.cu", 0, NULL, NULL) );
    const char *opts[] = {}; //{"--ptxas-options -v"}; //{"-rdc=true", "--ptxas-options -v"};
    hiprtcResult compileResult = hiprtcCompileProgram(prog, 0, opts); 

    size_t logSize2;
    NVRTC_SAFE_CALL("hiprtcGetProgramLogSize", hiprtcGetProgramLogSize(prog, &logSize2) );
    char *log = new char[logSize2];
    NVRTC_SAFE_CALL("hiprtcGetProgramLog", hiprtcGetProgramLog(prog, log) );
    std::cout << log << '\n';
    //std::err << log << '\n';
    delete[] log;

    if (compileResult != HIPRTC_SUCCESS) {
      cout << "failed to compile" << endl;
      return false;
    }

    size_t ptxSize;
    NVRTC_SAFE_CALL( "hiprtcGetCodeSize", hiprtcGetCodeSize(prog, &ptxSize) );
    char *ptx = new char[ptxSize];
    NVRTC_SAFE_CALL( "hiprtcGetCode", hiprtcGetCode(prog, ptx) );
    auto t1 = steady_clock::now();
    //cout << ptx << endl;

    auto t2 = steady_clock::now();
    hiprtcLinkState lState;
    
    hipJitOption options[6];
    void *optionVals[6];

    float walltime;
    char error_log[8192], info_log[8192];
    unsigned int logSize = 8192;

    // Setup linker options
    // Return walltime from JIT compilation
    options[0] = hipJitOptionWallTime;
    optionVals[0] = (void *)&walltime;
    // Pass a buffer for info messages
    options[1] = hipJitOptionInfoLogBuffer;
    optionVals[1] = (void *)info_log;
    // Pass the size of the info buffer
    options[2] = hipJitOptionInfoLogBufferSizeBytes;
    optionVals[2] = (void *)(long)logSize;
    // Pass a buffer for error message
    options[3] = hipJitOptionErrorLogBuffer;
    optionVals[3] = (void *)error_log;
    // Pass the size of the error buffer
    options[4] = hipJitOptionErrorLogBufferSizeBytes;
    optionVals[4] = (void *)(long)logSize;
    // Make the linker verbose
    options[5] = hipJitOptionLogVerbose;
    optionVals[5] = (void *)1;

    hiprtcLinkState *plState = &lState;
    checkCudaErrors( hiprtcLinkCreate(6, options, optionVals, plState) );
    //checkCudaErrors( hiprtcLinkCreate(0, 0, 0, plState) );
    checkCudaErrors( hiprtcLinkAddData(*plState, hipJitInputPtx, (void *)ptx, ptxSize+1, "device.ptx", 0, 0, 0) );
    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);
    
    delete ptx;

    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    //printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    void* cuOut;
    size_t outSize;
    hipError_t curesult = hiprtcLinkComplete(*plState, &cuOut, &outSize);

    printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    if (curesult != hipSuccess) {
        return false;
    }

    checkCudaErrors( hipModuleLoadData(&hModule, cuOut) );
    checkCudaErrors( hipModuleGetFunction(&hKernel, hModule, "kernel") );
    checkCudaErrors( hiprtcLinkDestroy(*plState) );

    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    //printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    auto t3 = steady_clock::now();

    cout << duration_cast<milliseconds>(t1-t0).count() << endl;
    cout << duration_cast<milliseconds>(t3-t2).count() << endl;
    return true;
}

void NvrtcContainer::run(const int numBlocks, const int numThreads) {
    int nThreads = numThreads;
    int nBlocks = numBlocks;
    dim3 block(nThreads, 1, 1);
    dim3 grid(nBlocks, 1, 1);
  
    void *args[0] = {};
  
    checkCudaErrors( hipModuleLaunchKernel(hKernel, grid.x, grid.y, grid.z, block.x, block.y, block.z, 0, NULL, args, NULL) );
    hipDeviceSynchronize();
}

NvrtcContainer::~NvrtcContainer() {
    if (hModule) {
        checkCudaErrors( hipModuleUnload(hModule) );
        hModule = 0;
    }
}