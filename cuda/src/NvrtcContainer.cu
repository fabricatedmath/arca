#include "hip/hip_runtime.h"
#include "NvrtcContainer.cuh"

#include <iostream>

#include <hip/hiprtc.h>

#define CUDA_DRIVER_API

#include <hip/hip_runtime_api.h>

using namespace std;

#define NVRTC_SAFE_CALL(Name, x)                                \
  do {                                                          \
    hiprtcResult result = x;                                     \
    if (result != HIPRTC_SUCCESS) {                              \
      std::cerr << "\nerror: " << Name << " failed with error " \
                << hiprtcGetErrorString(result);                 \
      exit(1);                                                  \
    }                                                           \
  } while (0)

NvrtcContainer::NvrtcContainer() : hModule(0), hKernel(0) {}

void NvrtcContainer::init() {
    checkCudaErrors( hipInit(0) );
}

bool NvrtcContainer::compile(const char* str, const int strlen) {
    hiprtcProgram prog;
    NVRTC_SAFE_CALL("hiprtcCreateProgram", hiprtcCreateProgram(&prog, str, "device.cu", 0, NULL, NULL) );
    const char *opts[] = {}; //{"--ptxas-options -v"}; //{"-rdc=true", "--ptxas-options -v"};
    hiprtcResult compileResult = hiprtcCompileProgram(prog, 0, opts); 

    size_t logSize2;
    NVRTC_SAFE_CALL("hiprtcGetProgramLogSize", hiprtcGetProgramLogSize(prog, &logSize2) );
    char *log = new char[logSize2];
    NVRTC_SAFE_CALL("hiprtcGetProgramLog", hiprtcGetProgramLog(prog, log) );
    std::cout << log << '\n';
    delete[] log;

    if (compileResult != HIPRTC_SUCCESS) {
      cout << "failed to compile" << endl;
      return false;
    }

    size_t ptxSize;
    NVRTC_SAFE_CALL( "hiprtcGetCodeSize", hiprtcGetCodeSize(prog, &ptxSize) );
    char *ptx = new char[ptxSize];
    NVRTC_SAFE_CALL( "hiprtcGetCode", hiprtcGetCode(prog, ptx) );
    cout << ptx << endl;

    hiprtcLinkState lState;
    hipJitOption options[6];
    void *optionVals[6];

    float walltime;
    char error_log[8192], info_log[8192];
    unsigned int logSize = 8192;

    // Setup linker options
    // Return walltime from JIT compilation
    options[0] = hipJitOptionWallTime;
    optionVals[0] = (void *)&walltime;
    // Pass a buffer for info messages
    options[1] = hipJitOptionInfoLogBuffer;
    optionVals[1] = (void *)info_log;
    // Pass the size of the info buffer
    options[2] = hipJitOptionInfoLogBufferSizeBytes;
    optionVals[2] = (void *)(long)logSize;
    // Pass a buffer for error message
    options[3] = hipJitOptionErrorLogBuffer;
    optionVals[3] = (void *)error_log;
    // Pass the size of the error buffer
    options[4] = hipJitOptionErrorLogBufferSizeBytes;
    optionVals[4] = (void *)(long)logSize;
    // Make the linker verbose
    options[5] = hipJitOptionLogVerbose;
    optionVals[5] = (void *)1;

    hiprtcLinkState *plState = &lState;

    //checkCudaErrors( hipInit(0) );
    //hipSetDevice(0);
    hipDevice_t cuDevice;
    checkCudaErrors( hipDeviceGet(&cuDevice, 0) );
    hipCtx_t context;
    checkCudaErrors( hipCtxCreate(&context, 0, cuDevice) );
    checkCudaErrors( hiprtcLinkCreate(6, options, optionVals, plState) );
    checkCudaErrors( hiprtcLinkAddData(*plState, hipJitInputPtx, (void *)ptx, ptxSize+1, "device.ptx", 0, 0, 0) );
    printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);
    
    delete ptx;

    printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    void* cuOut;
    size_t outSize;
    hipError_t curesult = hiprtcLinkComplete(*plState, &cuOut, &outSize);

    printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    if (curesult != hipSuccess) {
        return false;
    }

    checkCudaErrors( hipModuleLoadData(&hModule, cuOut) );
    checkCudaErrors( hipModuleGetFunction(&hKernel, hModule, "kernel") );
    checkCudaErrors( hiprtcLinkDestroy(*plState) );

    printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    return true;
}

void NvrtcContainer::run() {
    int nThreads = 32;
    int nBlocks = 1;
    dim3 block(nThreads, 1, 1);
    dim3 grid(nBlocks, 1, 1);
  
    void *args[0] = {};
  
    checkCudaErrors( hipModuleLaunchKernel(hKernel, grid.x, grid.y, grid.z, block.x, block.y, block.z, 0, NULL, args, NULL) );
    hipDeviceSynchronize();
}

NvrtcContainer::~NvrtcContainer() {
    if (hModule) {
        checkCudaErrors( hipModuleUnload(hModule) );
        hModule = 0;
    }
}