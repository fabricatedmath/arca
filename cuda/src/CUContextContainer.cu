#include "CUContextContainer.cuh"

#define CUDA_DRIVER_API

#include <hip/hip_runtime_api.h>

CUContextContainer::CUContextContainer() {
    checkCudaErrors( hipInit(0) );
    hipDevice_t cuDevice;
    checkCudaErrors( hipDeviceGet(&cuDevice, 0) );
    ctx = new hipCtx_t();
    checkCudaErrors( hipCtxCreate(ctx, 0, cuDevice) );
}

hipCtx_t* CUContextContainer::getCtx() {
    return ctx;
}

CUContextContainer::~CUContextContainer() {
    checkCudaErrors( hipCtxDestroy(*ctx) );
    delete ctx;
}