#include "CUContextContainer.cuh"

#define CUDA_DRIVER_API

#include <HelperCuda.h>

CUContextContainer::CUContextContainer() : CUContextContainer(0) {}

CUContextContainer::CUContextContainer(int deviceNum) {
    checkCudaErrors( hipInit(0) );
    hipDevice_t cuDevice;
    checkCudaErrors( hipDeviceGet(&cuDevice, deviceNum) );
    cuContext = new hipCtx_t();
    checkCudaErrors( hipCtxCreate(cuContext, 0, cuDevice) );
}

hipCtx_t* CUContextContainer::getCuContext() {
    return cuContext;
}

void CUContextContainer::setCurrentContext() {
    hipCtxSetCurrent(*cuContext);
}

void CUContextContainer::popContext() {
    hipCtxPopCurrent(NULL);
}

CUContextContainer::~CUContextContainer() {
    checkCudaErrors( hipCtxDestroy(*cuContext) );
    delete cuContext;
}