#include "CUContextContainer.cuh"

#define CUDA_DRIVER_API

#include <HelperCuda.h>

CUContextContainer::CUContextContainer() {
    checkCudaErrors( hipInit(0) );
    hipDevice_t cuDevice;
    checkCudaErrors( hipDeviceGet(&cuDevice, 0) );
    cuContext = new hipCtx_t();
    checkCudaErrors( hipCtxCreate(cuContext, 0, cuDevice) );
}

hipCtx_t* CUContextContainer::getCuContext() {
    return cuContext;
}

void CUContextContainer::setCurrentContext() {
    hipCtxSetCurrent(*cuContext);
}

void CUContextContainer::popContext() {
    hipCtxPopCurrent(NULL);
}

CUContextContainer::~CUContextContainer() {
    checkCudaErrors( hipCtxDestroy(*cuContext) );
    delete cuContext;
}