#include "PtxCompiler.cuh"

#include <iostream>

using namespace std;

#include <hip/hiprtc.h>

#define NVRTC_SAFE_CALL(Name, x)                                \
  do {                                                          \
    hiprtcResult result = x;                                     \
    if (result != HIPRTC_SUCCESS) {                              \
      std::cerr << "\nerror: " << Name << " failed with error " \
                << hiprtcGetErrorString(result);                 \
      exit(1);                                                  \
    }                                                           \
  } while (0)

PtxCompiler::PtxCompiler() : ptxStr(NULL), ptxStrLen(0), logStr(NULL), logStrLen(0) {}

int PtxCompiler::compile(const char* str, const int strLen, bool rdc) {
    const string progStr(str,strLen);
    hiprtcProgram prog;
    NVRTC_SAFE_CALL("hiprtcCreateProgram", hiprtcCreateProgram(&prog, progStr.c_str(), NULL, 0, NULL, NULL) );
    hiprtcResult compileResult;

    if (rdc) {
        const char *opts[] = {"-rdc=true", "--gpu-architecture=compute_75"};
        compileResult = hiprtcCompileProgram(prog, 2, opts); 
    } else {
        const char *opts[] = {"--gpu-architecture=compute_75"};
        compileResult = hiprtcCompileProgram(prog, 1, opts); 
    }

    NVRTC_SAFE_CALL("hiprtcGetProgramLogSize", hiprtcGetProgramLogSize(prog, &logStrLen) );
    logStr = new char[logStrLen];
    NVRTC_SAFE_CALL("hiprtcGetProgramLog", hiprtcGetProgramLog(prog, logStr) );

    if (compileResult != HIPRTC_SUCCESS) {
        return compileResult;
    }

    NVRTC_SAFE_CALL( "hiprtcGetCodeSize", hiprtcGetCodeSize(prog, &ptxStrLen) );
    ptxStr = new char[ptxStrLen];
    NVRTC_SAFE_CALL( "hiprtcGetCode", hiprtcGetCode(prog, ptxStr) );
    return compileResult;
}

char* PtxCompiler::getPtxStr() {
    return ptxStr;
}

size_t PtxCompiler::getPtxStrLen() {
    return ptxStrLen;
}

char* PtxCompiler::getLogStr() {
    return logStr;
}

size_t PtxCompiler::getLogStrLen() {
    return logStrLen;
}

PtxCompiler::~PtxCompiler() {
    if (ptxStr != NULL) {
        delete ptxStr;
    }

    if (logStr != NULL) {
        delete logStr;
    }
}