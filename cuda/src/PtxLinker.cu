#include "PtxLinker.cuh"

#include <iostream>

using namespace std;

#define CUDA_DRIVER_API

#include <hip/hip_runtime_api.h>

#include <chrono>
using namespace std::chrono;

#define CURESULT_SAFE_CALL(x)                                \
  do {                                                          \
    hipError_t result = x;                                     \
    if (result != hipSuccess) {                              \
      return result;                                                  \
    }                                                           \
  } while (0)

PtxLinker::PtxLinker() 
    : hModule(0), hKernel(0) {}

int PtxLinker::link(const char* ptx, const int ptxLen, const char* funcNameStr, const int funcNameStrLen) {
    auto t0 = steady_clock::now();
    const string funcName(funcNameStr,funcNameStrLen);

    hiprtcLinkState lState;
    
    hipJitOption options[6];
    void *optionVals[6];

    float walltime;
    char error_log[8192], info_log[8192];
    unsigned int logSize = 8192;

    // Setup linker options
    // Return walltime from JIT compilation
    options[0] = hipJitOptionWallTime;
    optionVals[0] = (void *)&walltime;
    // Pass a buffer for info messages
    options[1] = hipJitOptionInfoLogBuffer;
    optionVals[1] = (void *)info_log;
    // Pass the size of the info buffer
    options[2] = hipJitOptionInfoLogBufferSizeBytes;
    optionVals[2] = (void *)(long)logSize;
    // Pass a buffer for error message
    options[3] = hipJitOptionErrorLogBuffer;
    optionVals[3] = (void *)error_log;
    // Pass the size of the error buffer
    options[4] = hipJitOptionErrorLogBufferSizeBytes;
    optionVals[4] = (void *)(long)logSize;
    // Make the linker verbose
    options[5] = hipJitOptionLogVerbose;
    optionVals[5] = (void *)1;

    hiprtcLinkState *plState = &lState;
    CURESULT_SAFE_CALL( hiprtcLinkCreate(6, options, optionVals, plState) );
    //checkCudaErrors( hiprtcLinkCreate(0, 0, 0, plState) );
    CURESULT_SAFE_CALL( hiprtcLinkAddData(*plState, hipJitInputPtx, (void *)ptx, ptxLen, NULL, 0, 0, 0) );
    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    //printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    void* cuOut;
    size_t outSize;
    hipError_t curesult = hiprtcLinkComplete(*plState, &cuOut, &outSize);

    printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    if (curesult != hipSuccess) {
        return curesult;
    }

    CURESULT_SAFE_CALL( hipModuleLoadData(&hModule, cuOut) );
    CURESULT_SAFE_CALL( hipModuleGetFunction(&hKernel, hModule, funcName.c_str()) );
    CURESULT_SAFE_CALL( hiprtcLinkDestroy(*plState) );

    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    //printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    auto t1 = steady_clock::now();

    cout << duration_cast<milliseconds>(t1-t0).count() << endl;
    return hipSuccess;
}

int PtxLinker::run(const int numBlocks, const int numThreads) {
    int nThreads = numThreads;
    int nBlocks = numBlocks;
    dim3 block(nThreads, 1, 1);
    dim3 grid(nBlocks, 1, 1);
  
    void *args[0] = {};
  
    CURESULT_SAFE_CALL( hipModuleLaunchKernel(hKernel, grid.x, grid.y, grid.z, block.x, block.y, block.z, 0, NULL, args, NULL) );
    hipDeviceSynchronize();
    
    return hipSuccess;
}

PtxLinker::~PtxLinker() {
    if (hModule) {
        checkCudaErrors( hipModuleUnload(hModule) );
        hModule = 0;
    }
}