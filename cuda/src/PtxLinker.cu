#include "PtxLinker.cuh"

#include <iostream>

using namespace std;

#define CUDA_DRIVER_API

#include <HelperCuda.h>

#include <chrono>
using namespace std::chrono;

#define CURESULT_SAFE_CALL(x)         \
  do {                                \
    hipError_t result = x;              \
    if (result != hipSuccess) {     \
      return result;                  \
    }                                 \
  } while (0)

PtxLinker::PtxLinker() 
    : hModule(0)
    , hKernel(0)
    , infoLogStr(new char[logSize])
    , infoLogStrLen(0)
    , errorLogStr(new char[logSize])
    , errorLogStrLen(0) {}

int PtxLinker::link(const char* ptx, const int ptxLen, const char* funcNameStr, const int funcNameStrLen) {
    const string funcName(funcNameStr,funcNameStrLen);

    hiprtcLinkState lState;
    
    hipJitOption options[6];
    void *optionVals[6];

    float walltime;
    //char error_log[8192], info_log[8192];
    //unsigned int logSize = 8192;

    // Setup linker options
    // Return walltime from JIT compilation
    options[0] = hipJitOptionWallTime;
    optionVals[0] = (void *)&walltime;
    // Pass a buffer for info messages
    options[1] = hipJitOptionInfoLogBuffer;
    optionVals[1] = (void *)infoLogStr;
    // Pass the size of the info buffer
    options[2] = hipJitOptionInfoLogBufferSizeBytes;
    optionVals[2] = (void *)(long)logSize;
    // Pass a buffer for error message
    options[3] = hipJitOptionErrorLogBuffer;
    optionVals[3] = (void *)errorLogStr;
    // Pass the size of the error buffer
    options[4] = hipJitOptionErrorLogBufferSizeBytes;
    optionVals[4] = (void *)(long)logSize;
    // Make the linker verbose
    options[5] = hipJitOptionLogVerbose;
    optionVals[5] = (void *)1;

    CURESULT_SAFE_CALL( hiprtcLinkCreate(6, options, optionVals, &lState) );
    infoLogStrLen = logSize;
    errorLogStrLen = logSize;

    CURESULT_SAFE_CALL( hiprtcLinkAddData(lState, hipJitInputPtx, (void *)ptx, ptxLen, NULL, 0, 0, 0) );

    void* cuOut;
    size_t outSize;
    CURESULT_SAFE_CALL( hiprtcLinkComplete(lState, &cuOut, &outSize) );
    CURESULT_SAFE_CALL( hipModuleLoadData(&hModule, cuOut) );
    CURESULT_SAFE_CALL( hipModuleGetFunction(&hKernel, hModule, funcName.c_str()) );
    CURESULT_SAFE_CALL( hiprtcLinkDestroy(lState) );
    return hipSuccess;
}

int PtxLinker::run(const int numBlocks, const int numThreads) {
    int nThreads = numThreads;
    int nBlocks = numBlocks;
    dim3 block(nThreads, 1, 1);
    dim3 grid(nBlocks, 1, 1);
  
    void *args[0] = {};
  
    CURESULT_SAFE_CALL( hipModuleLaunchKernel(hKernel, grid.x, grid.y, grid.z, block.x, block.y, block.z, 0, NULL, args, NULL) );
    hipDeviceSynchronize();

    return hipSuccess;
}

char* PtxLinker::getInfoLogStr() {
    return infoLogStr;
}

size_t PtxLinker::getInfoLogStrLen() {
    return infoLogStrLen;
}

char* PtxLinker::getErrorLogStr() {
    return errorLogStr;
}

size_t PtxLinker::getErrorLogStrLen() {
    return errorLogStrLen;
}

PtxLinker::~PtxLinker() {
    if (hModule) {
        checkCudaErrors( hipModuleUnload(hModule) );
        hModule = 0;
    }
    delete infoLogStr;
    delete errorLogStr;
}