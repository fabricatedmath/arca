#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

using namespace std;

#include "global.cuh"

#define CUDA_DRIVER_API

#include <hip/hip_runtime_api.h>

#define NVRTC_SAFE_CALL(Name, x)                                \
  do {                                                          \
    hiprtcResult result = x;                                     \
    if (result != HIPRTC_SUCCESS) {                              \
      std::cerr << "\nerror: " << Name << " failed with error " \
                << hiprtcGetErrorString(result);                 \
      exit(1);                                                  \
    }                                                           \
  } while (0)

const char *saxpy = "                                           \n\
 __device__                                         \n\
int device_call()                                               \n\
{                                                               \n\
    if (threadIdx.x == 0) {                                     \n\
        printf(\"llama\\n\");                                     \n\
    }                                                           \n\
    return 7;                                                    \n\
  } \n\
                                                              \n";

int main() {
  //call();
    hiprtcProgram prog;
    NVRTC_SAFE_CALL("hiprtcCreateProgram", hiprtcCreateProgram(&prog, saxpy, "device.cu", 0, NULL, NULL) );
    const char *opts[] = {"-rdc=true"};
    hiprtcResult compileResult = hiprtcCompileProgram(prog, 1, opts); 

    size_t logSize2;
    NVRTC_SAFE_CALL("hiprtcGetProgramLogSize", hiprtcGetProgramLogSize(prog, &logSize2) );
    char *log = new char[logSize2];
    NVRTC_SAFE_CALL("hiprtcGetProgramLog", hiprtcGetProgramLog(prog, log) );
    std::cout << log << '\n';
    delete[] log;

    if (compileResult != HIPRTC_SUCCESS) {
      exit(1);
    }

    size_t ptxSize;
    NVRTC_SAFE_CALL( "hiprtcGetCodeSize", hiprtcGetCodeSize(prog, &ptxSize) );
    char *ptx = new char[ptxSize];
    NVRTC_SAFE_CALL( "hiprtcGetCode", hiprtcGetCode(prog, ptx) );
    //cout << ptx << endl;

    hiprtcLinkState lState;
    hipJitOption options[6];
    void *optionVals[6];

    float walltime;
    char error_log[8192], info_log[8192];
    unsigned int logSize = 8192;

    // Setup linker options
    // Return walltime from JIT compilation
    options[0] = hipJitOptionWallTime;
    optionVals[0] = (void *)&walltime;
    // Pass a buffer for info messages
    options[1] = hipJitOptionInfoLogBuffer;
    optionVals[1] = (void *)info_log;
    // Pass the size of the info buffer
    options[2] = hipJitOptionInfoLogBufferSizeBytes;
    optionVals[2] = (void *)(long)logSize;
    // Pass a buffer for error message
    options[3] = hipJitOptionErrorLogBuffer;
    optionVals[3] = (void *)error_log;
    // Pass the size of the error buffer
    options[4] = hipJitOptionErrorLogBufferSizeBytes;
    optionVals[4] = (void *)(long)logSize;
    // Make the linker verbose
    options[5] = hipJitOptionLogVerbose;
    optionVals[5] = (void *)1;

    hiprtcLinkState *plState = &lState;

    checkCudaErrors( hipInit(0) );
    //hipSetDevice(0);
    hipDevice_t cuDevice;
    checkCudaErrors( hipDeviceGet(&cuDevice, 0) );
    hipCtx_t context;
    checkCudaErrors( hipCtxCreate(&context, 0, cuDevice) );
    checkCudaErrors( hiprtcLinkCreate(6, options, optionVals, plState) );
    checkCudaErrors( hiprtcLinkAddData(*plState, hipJitInputPtx, (void *)ptx, ptxSize+1, "device.ptx", 0, 0, 0) );
    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);
    checkCudaErrors( hiprtcLinkAddFile(*plState, hipJitInputPtx, "build/global.ptx", 0, 0, 0) );
    //checkCudaErrors( hiprtcLinkAddFile(*plState, hipJitInputPtx, "build/device.ptx", 0, 0, 0) );
    


    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    //printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    void* cuOut;
    size_t outSize;
    hipError_t curesult = hiprtcLinkComplete(*plState, &cuOut, &outSize);

    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    //printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    if (curesult != hipSuccess) {
      exit(1);
    }

    hipModule_t hModule = 0;
    hipFunction_t hKernel = 0;

    hipModule_t* phModule = &hModule;
    hipFunction_t* phKernel = &hKernel;

    checkCudaErrors( hipModuleLoadData(phModule, cuOut) );
    checkCudaErrors( hipModuleGetFunction(phKernel, *phModule, "kernel") );
    checkCudaErrors( hiprtcLinkDestroy(*plState) );

    //printf("CUDA Link Completed in %fms. Linker Output:\n%s\n", walltime, info_log);

    //printf("CUDA Link Completed in %fms. Linker Error Output:\n%s\n", walltime, error_log);

    int nThreads = 32;
    int nBlocks = 1;
    dim3 block(nThreads, 1, 1);
    dim3 grid(nBlocks, 1, 1);
  
    void *args[0] = {};
  
    checkCudaErrors( hipModuleLaunchKernel(hKernel, grid.x, grid.y, grid.z, block.x, block.y, block.z, 0, NULL, args, NULL) );
    hipDeviceSynchronize();

    if (hModule) {
        checkCudaErrors( hipModuleUnload(hModule) );
        hModule = 0;
    }
}